#include "hip/hip_runtime.h"
#include "HashTable.cuh"
#include <bitset>

#define ADDRESS_LANE 31

__device__ uint32_t Slab::ReadSlab(Address slab_addr, int laneID) {
	return *(SlabAddress(slab_addr, laneID));
}

HashTable::HashTable(int size, SlabAlloc * s) {
	no_of_buckets = size;
	slab_alloc = s;
	hipMalloc(&base_slabs, no_of_buckets*sizeof(Address));
	int threads_per_block = 32 /* warp size */ , blocks = no_of_buckets;
	init<<<blocks, threads_per_block>>>();
}

__global__ void HashTable::init() {
	ResidentBlock rb;	rb.init(slab_alloc);
	int i = blockIdx.x;
	while (i < no_of_buckets) {
		base_slabs[i] = rb.warp_allocate();
		i += gridDim.x;
	}
}

__device__ void HashTableOperation::init(HashTable * h, ResidentBlock * rb, Instruction ins) {
	hashtable = h;
	resident_block = rb;
	instr = ins;
	is_active = true;
	std::bitset<32> valid_key_mask(std::string("10101010101010101010101010101000"));
	VALID_KEY_MASK = valid_key_mask.to_ulong();
	WARP_MASK = (1llu << 32) - 1;
}

__device__ void HashTableOperation::run() {
	auto work_queue = __ballot(WARP_MASK, is_active), old_work_queue = 0;
	while(work_queue != 0) {
		src_lane = __ffs(work_queue);
		assert(src_lane>=1 && src_lane <= 32);
		--src_lane;
		Instruction::Type src_instrtype = __shfl_sync(WARP_MASK, instr.type, src_lane);
		src_key = __shfl_sync(WARP_MASK, instr.key, src_lane);
		src_value = __shfl_sync(WARP_MASK, instr.value, src_lane);
		unsigned src_bucket = HashFunction::hash(src_key, hashtable->no_of_buckets);
		if(work_queue != old_work_queue) {
			next = hashtable->base_slabs[src_bucket];
		}
		read_data = Slab::ReadSlab(next, laneID);
		switch(src_instrtype) {
			case Instruction::Insert:
				inserter();
				break;
			case Instruction::Delete:
				deleter();
				break;
			case Instruction::Search:
				searcher();
				break;
		}
		old_work_queue = work_queue;
		work_queue = __ballot(WARP_MASK, is_active);
	}
}
