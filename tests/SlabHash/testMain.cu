#include "hip/hip_runtime.h"
#include "SlabHash/HashTable.cuh"
#include "errorcheck.h"
#include <cstdio>
#include <assert.h>

__device__ inline int laneID() {
	return threadIdx.x % warpSize;
}

__device__ inline int warpID() {
	return CEILDIV(blockDim.x, warpSize) * blockIdx.x + (threadIdx.x / warpSize);
}

__device__ void readanddeallocate(SlabAlloc * s, ResidentBlock * rb, Address a) {
	uint32_t data1 = *(s->SlabAddress(a, laneID()));
	__syncwarp();
	Address address = __shfl_sync(WARP_MASK, data1, ADDRESS_LANE);
	uint32_t data2 = *(s->SlabAddress(address, laneID()));
	//s->deallocate(a);
	//s->deallocate(address);
	if(laneID() != 31 && (data1 != warpID() || data2 != warpID() + (1 << 18)))
		printf("After writing, Warp %d, Lane %d: Slab 1 - %d, Slab 2 - %d\n", warpID(), laneID(), data1, data2);
}

__device__ float sum_local_rbl_changes = 0.0;
__device__ float sum_sqr_local_rbl_changes = 0.0;

__global__ void checkallbitmaps(SlabAlloc* s) {
	//Checking if array s->bitmaps has been copied properly (it most probably has)
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	while (i < s->maxSuperBlocks * SuperBlock::numMemoryBlocks) {
		uint32_t Bitmap = s->bitmaps[i / 32].bitmap[i % 32];
		if (Bitmap != 0) {
			printf("s->bitmaps[%d].bitmap[%d] = %x, instead of 0\n", i / 32, i % 32, Bitmap);
		}
		i += gridDim.x;
	}
}

__global__ void kernel(SlabAlloc * s) {
	ResidentBlock rb(s);
	int x = 0;
	int y = 0;

	Address a = rb.warp_allocate(&x), a2 = rb.warp_allocate(&y);
	
	// Calculation of average local_rbl_changes, and terminating threads for whom any one warp_allocate() fails
	float avg = ((float)x + (float)y) / 2;
	atomicAdd(&sum_local_rbl_changes, avg);
	atomicAdd(&sum_sqr_local_rbl_changes, (avg*avg));
	if (a == EMPTY_ADDRESS || a2 == EMPTY_ADDRESS) {
		if (a != EMPTY_ADDRESS)	s->deallocate(a);
		else if (a2 != EMPTY_ADDRESS)	s->deallocate(a2);
		return;
	}

	// Checking if all slabs have been initialized properly
	uint32_t data1 = *(s->SlabAddress(a, laneID())), data2 = *(s->SlabAddress(a2, laneID()));
	if((data1 != 0xFFFFFFFF || data2 != 0xFFFFFFFF))
		printf("Before writing, Warp %d, Lane %d: Slab 1 - %x, Slab 2 - %x\n", warpID(), laneID(), data1, data2);

	auto ptr = s->SlabAddress(a, laneID());
	*ptr = warpID();
	if(laneID() == ADDRESS_LANE) {
		*ptr = a2;
	}
	ptr = s->SlabAddress(a2, laneID());
	*ptr = warpID()+(1<<18);
	
	readanddeallocate(s, &rb, a);
}

void test1() {
	const ULL log2slabsPerWarp = 0;	// Cannot be greater than SLAB_BITS(10) + MEMORYBLOCK_BITS(8)
	// Make sure numWarps is big enough so that numSuperBlocks is non-zero
	const ULL numWarps = 1 << 18, numSuperBlocks = numWarps >> SLAB_BITS + MEMORYBLOCK_BITS - log2slabsPerWarp;
	SlabAlloc * s = new SlabAlloc(numSuperBlocks);
	SlabAlloc * d_s;
	gpuErrchk(hipMalloc(&d_s, sizeof(SlabAlloc)));
	gpuErrchk(hipMemcpy(d_s, s, sizeof(SlabAlloc), hipMemcpyDefault));
	int numBlocks = numWarps>>5, threadsPerBlock = 1024;
	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, 1<<28));

	/*checkallbitmaps <<< ((s->maxSuperBlocks * SuperBlock::numMemoryBlocks) >> 5), 1024 >>> (d_s);
	gpuErrchk(hipDeviceSynchronize());
	printf("Completed check of array s->bitmaps before running kernel\n");*/

	kernel<<<numBlocks,threadsPerBlock>>>(d_s);
	gpuErrchk(hipDeviceSynchronize());

	/*checkallbitmaps <<< ((s->maxSuperBlocks * SuperBlock::numMemoryBlocks) >> 5), 1024 >>> (d_s);
	gpuErrchk(hipDeviceSynchronize());
	printf("Completed check of array s->bitmaps after running kernel\n");*/

	float avg_local_rbl_changes = 0.0, var_local_rbl_changes = 0.0;
	gpuErrchk(hipMemcpyFromSymbol(&avg_local_rbl_changes, HIP_SYMBOL(sum_local_rbl_changes), sizeof(float)));
	gpuErrchk(hipMemcpyFromSymbol(&var_local_rbl_changes, HIP_SYMBOL(sum_sqr_local_rbl_changes), sizeof(float)));
	avg_local_rbl_changes /= (numWarps << 5);
	var_local_rbl_changes = var_local_rbl_changes / (numWarps << 5) - (avg_local_rbl_changes * avg_local_rbl_changes);
	printf("Average local_rbl_changes = %f, Variance in local_rbl_changes=%f\n", avg_local_rbl_changes, var_local_rbl_changes);

	gpuErrchk(hipMemcpy(s, d_s, sizeof(SuperBlock), hipMemcpyDefault));
	printf("Final no. of superblocks: %d\n", s->getNumSuperBlocks());
	gpuErrchk(hipFree(d_s));
	delete s;
}


__global__ void kernel2(SlabAlloc * s) {
	ResidentBlock rb(s);
	int x = 0;
	for(int i = 0; i <= MemoryBlock::numSlabs; ++i) {
		if(threadIdx.x == 0)	printf("\r%.4d", i);
		rb.warp_allocate(&x);
	}
	s->allocateSuperBlock();
}

void test2() {
	const ULL numWarps = 1, numSuperBlocks = 1;
	SlabAlloc * s = new SlabAlloc(numSuperBlocks);
	SlabAlloc * d_s;
	gpuErrchk(hipMalloc(&d_s, sizeof(SlabAlloc)));
	gpuErrchk(hipMemcpy(d_s, s, sizeof(SlabAlloc), hipMemcpyDefault));
	int numBlocks = numWarps, threadsPerBlock = 32;
	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, 1<<28));

	kernel2<<<numBlocks,threadsPerBlock>>>(d_s);

	gpuErrchk(hipFree(d_s));
}


__managed__ uint32_t search_success = 0;
__managed__ uint32_t delete_success = 0;
__managed__ uint32_t finder_success = 0;

__device__ inline uint32_t key() {		return blockIdx.x;	}
__device__ inline uint32_t value() {	return threadIdx.x;	}

__global__ void kernel3ins(HashTable* h, SlabAlloc* s) {
	ResidentBlock rb(s);
	uint32_t key = key(), value = value();
	Instruction ins;
	ins.type = Instruction::Type::Insert;
	ins.key = key;
	ins.value = value;
	HashTableOperation op(&ins, h, &rb);
	op.run();
}

__global__ void kernel3inscheck(HashTable* h, SlabAlloc* s) {
	ResidentBlock rb(s);
	uint32_t key = key(), value = value();
	Instruction ins;
	ins.type = Instruction::Type::Search;
	ins.key = key;
	ins.value = SEARCH_NOT_FOUND;
	HashTableOperation op(&ins, h, &rb);
	op.run();
	if (ins.value != SEARCH_NOT_FOUND) {
		atomicAdd(&search_success, 1);
	}
}

__global__ void kernel3find(HashTable* h, SlabAlloc* s) {
	ResidentBlock rb(s);
	uint32_t key = key(), value = value();
	Instruction ins;
	ins.type = Instruction::Type::FindAll;
	ins.key = key;
	HashTableOperation op(&ins, h, &rb, threadIdx.x == 0);
	__syncwarp();
	op.run();
	if (threadIdx.x == 0 && ins.no_of_found_values == blockDim.x) {
		if (ins.findererror == 0) {
			if (ins.foundvalues[0] < blockDim.x && ins.foundvalues[1] < blockDim.x) {
				atomicAdd(&finder_success, 1);
			}
		}
	}
}

__global__ void kernel3del(HashTable* h, SlabAlloc* s) {
	ResidentBlock rb(s);
	uint32_t key = key(), value = value();
	Instruction ins;
	ins.type = Instruction::Type::Delete;
	ins.key = key;
	ins.value = value;
	HashTableOperation op(&ins, h, &rb);
	op.run();
}

__global__ void kernel3delcheck(HashTable* h, SlabAlloc* s) {
	ResidentBlock rb(s);
	uint32_t key = key(), value = value();
	Instruction ins;
	ins.type = Instruction::Type::Search;
	ins.key = key;
	ins.value = SEARCH_NOT_FOUND;
	HashTableOperation op(&ins, h, &rb);
	op.run();
	if (ins.value == SEARCH_NOT_FOUND) {
		atomicAdd(&delete_success, 1);
	}
}

void test3() {
	const ULL numThreads = 1<<18;
	const ULL numSuperBlocks = 1, numWarps = numThreads >> 5;
	SlabAlloc * s = new SlabAlloc(numSuperBlocks);
	SlabAlloc * d_s;
	gpuErrchk(hipMalloc(&d_s, sizeof(SlabAlloc)));
	gpuErrchk(hipMemcpy(d_s, s, sizeof(SlabAlloc), hipMemcpyDefault));
	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, 1<<30));
	
	int no_of_buckets = numThreads / 128;	// avg slabs per bucket ~ 9-10, assuming 1 insert instruction per thread
	HashTable * h = new HashTable(no_of_buckets, d_s);
	HashTable * d_h;
	gpuErrchk(hipMalloc(&d_h, sizeof(HashTable)));
	gpuErrchk(hipMemcpy(d_h, h, sizeof(HashTable), hipMemcpyDefault));
	
	int numBlocks = numWarps>>5, threadsPerBlock = 1024;
	kernel3ins<<<numBlocks, threadsPerBlock>>>(d_h, d_s);
	kernel3inscheck<<<numBlocks, threadsPerBlock>>>(d_h, d_s);
	kernel3find<<<numBlocks, threadsPerBlock>>>(d_h, d_s);
	kernel3del<<<numBlocks, threadsPerBlock>>>(d_h, d_s);
	kernel3delcheck<<<numBlocks, threadsPerBlock>>>(d_h, d_s);

	gpuErrchk(hipFree(d_h));
	delete h;
	gpuErrchk(hipFree(d_s));
	delete s;

	printf("searcher() success rate = %f%\n", (float)search_success * 100 / (float)numThreads);
	printf("deleter() success rate = %f%\n", (float)delete_success * 100 / (float)numThreads);
	printf("finder() success rate = %f%\n", (float)finder_success*100/((float)numThreads/threadsPerBlock));
}


__global__ void kernel4(SlabAlloc* s) {
	ResidentBlock rb(s);
	Address a = rb.warp_allocate();
	uint32_t left = threadIdx.x, right = threadIdx.x + 5;
	uint32_t data[2] = { left, right };
	if (1 << laneID() & VALID_KEY_MASK) {
		*(ULL*)(s->SlabAddress(a, laneID())) = *reinterpret_cast<ULL *>(data);
		//assert(atomicCAS((ULL*)(s->SlabAddress(a, laneID())), (ULL)0xFFFFFFFFFFFFFFFF, *((ULL*)data)) == (ULL)0xFFFFFFFFFFFFFFFF);
		assert(*(s->SlabAddress(a, laneID())) == left);
		assert(*(s->SlabAddress(a, laneID()+1)) == right);
	}
}

void test4() {
	const ULL numWarps = 1, numSuperBlocks = 1;
	SlabAlloc * s = new SlabAlloc(numSuperBlocks);
	SlabAlloc * d_s;
	gpuErrchk(hipMalloc(&d_s, sizeof(SlabAlloc)));
	gpuErrchk(hipMemcpy(d_s, s, sizeof(SlabAlloc), hipMemcpyDefault));
	int numBlocks = numWarps, threadsPerBlock = 32;
	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, 1<<28));

	kernel4<<<numBlocks,threadsPerBlock>>>(d_s);

	gpuErrchk(hipFree(d_s));
}

int main() {
	test3();
	gpuErrchk(hipDeviceReset());
}
