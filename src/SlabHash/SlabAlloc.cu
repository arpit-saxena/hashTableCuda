#include "hip/hip_runtime.h"
#include "SlabAlloc.cuh"
#include "HashFunction.cuh"
#include <stdio.h>
#include <assert.h>
#include <new>

BlockBitMap::BlockBitMap() {
	memset(bitmap, 0, 32*sizeof(uint32_t));
}

__host__ __device__ Slab::Slab() {
	memset(arr, 0xFF, 32*sizeof(uint32_t));
}

__host__ SlabAlloc::SlabAlloc(int numSuperBlocks = maxSuperBlocks) : initNumSuperBlocks(numSuperBlocks) {
	this -> numSuperBlocks = numSuperBlocks;
	if (numSuperBlocks > maxSuperBlocks) {
		//TODO: Better way to handle this?
		printf("Can't allocate %d super blocks. Max is %d", numSuperBlocks, maxSuperBlocks);
		return;
	}

	hipMalloc(&superBlocks, maxSuperBlocks*sizeof(SuperBlock *));

	SuperBlock * sb = new SuperBlock();
	for (int i = 0; i < maxSuperBlocks; i++) {
		SuperBlock * temp = nullptr;
		if(i < numSuperBlocks) {
			hipMalloc(&temp, sizeof(SuperBlock));
			hipMemcpy(temp, sb , sizeof(SuperBlock), hipMemcpyDefault);
		}
		hipMemcpy(superBlocks + i, &temp, sizeof(SuperBlock *), hipMemcpyDefault);
	}
	delete sb;
}

__host__ SlabAlloc::~SlabAlloc() {
	int size = maxSuperBlocks - initNumSuperBlocks;
	if (size != 0) {
		int threadsPerBlock = 64, numBlocks = CEILDIV(size, threadsPerBlock);
		utilitykernel::clean_superblocks<<<numBlocks, threadsPerBlock>>>(superBlocks + initNumSuperBlocks, size);
	}

	SuperBlock **  h_superBlocks = new SuperBlock *[initNumSuperBlocks];
	hipMemcpy(h_superBlocks, superBlocks, initNumSuperBlocks*sizeof(SuperBlock *), hipMemcpyDefault);
	for (int i = 0; i < initNumSuperBlocks; i++) {
		if(h_superBlocks[i])	hipFree(h_superBlocks[i]);
	}
	delete h_superBlocks;
	hipFree(superBlocks);
}

__global__
void utilitykernel::clean_superblocks(SuperBlock ** superBlocks, const ULL size) {
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;
	while(threadID < size) {
		if(superBlocks[threadID])	free(superBlocks[threadID]);
		superBlocks[threadID] = nullptr;
		threadID += gridDim.x * blockDim.x;
	}
}

__device__ __host__
int SlabAlloc::getNumSuperBlocks() {
	return numSuperBlocks;
}

__device__
Address SlabAlloc::makeAddress(uint32_t superBlock_idx, uint32_t memoryBlock_idx, uint32_t slab_idx) {
	return (superBlock_idx << (SLAB_BITS + MEMORYBLOCK_BITS))
			+ (memoryBlock_idx << SLAB_BITS)
			+ slab_idx;
}

// Currently called with full warp only, so it also assumes full warp
__device__ int SlabAlloc::allocateSuperBlock() {
	assert(__activemask() == WARP_MASK);
	int workerThreadIdx = 0;
	int localIdx = -1;
	if (threadIdx.x % 32 == workerThreadIdx) {
		int numSuper = numSuperBlocks; // Get a local copy of the variable
		if (numSuper == maxSuperBlocks) {
			localIdx = numSuper - 1; // This is the last super block, deal with it
		} else {
			localIdx = numSuper++;
			SuperBlock * newSuperBlock = (SuperBlock *) malloc(sizeof(SuperBlock));
			SuperBlock * oldSuperBlock = (SuperBlock *) atomicCAS((ULL *) (superBlocks + localIdx), (ULL) nullptr, (ULL) newSuperBlock);
			if (oldSuperBlock != nullptr) {
				free(newSuperBlock);
			} else {
				atomicAdd(&numSuperBlocks, 1);
				new(newSuperBlock) SuperBlock();
			}
		}
	}

	__syncwarp();
	return __shfl_sync(WARP_MASK, localIdx, workerThreadIdx);
}

__device__ uint32_t * SlabAlloc::SlabAddress(Address addr, uint32_t laneID){
	uint32_t slab_idx = addr & ((1 << SLAB_BITS) - 1);
	uint32_t block_idx = (addr >> SLAB_BITS) & ((1 << MEMORYBLOCK_BITS) - 1);
	uint32_t superBlock_idx = (addr >> (SLAB_BITS + MEMORYBLOCK_BITS));
	return (superBlocks[superBlock_idx]->memoryBlocks[block_idx].slabs[slab_idx].arr) + laneID;
}

__device__ void SlabAlloc::deallocate(Address addr){		//Doesn't need a full warp
	unsigned global_memory_block_no = addr >> SLAB_BITS;
	unsigned memory_unit_no = addr & ((1<<SLAB_BITS)-1);		//addr%1024, basically
	unsigned lane_no = memory_unit_no / 32, slab_no = memory_unit_no % 32;
	int laneID = threadIdx.x % warpSize;
	if(laneID == __ffs(__activemask()) - 1){
		wipeSlab(addr);
		BlockBitMap * resident_bitmap = bitmaps + global_memory_block_no;
		uint32_t * global_bitmap_line = resident_bitmap->bitmap + lane_no;
		atomicAnd(global_bitmap_line, ~(1u << slab_no));
	}
	// TODO Check for divergence here
}

__device__ void SlabAlloc::wipeSlab(Address addr) {			//Doesn't need a full warp
	uint32_t slab_idx = addr & ((1 << SLAB_BITS) - 1);
	uint32_t block_idx = (addr >> SLAB_BITS) & ((1 << MEMORYBLOCK_BITS) - 1);
	uint32_t superBlock_idx = (addr >> (SLAB_BITS + MEMORYBLOCK_BITS));
	auto slabarr = superBlocks[superBlock_idx]->memoryBlocks[block_idx].slabs[slab_idx].arr;
	if (threadIdx.x % warpSize == __ffs(__activemask()) - 1) {
		memset(slabarr, 0xFF, 32 * sizeof(uint32_t));
	}
}

__device__ ResidentBlock::ResidentBlock(SlabAlloc * s) {
	slab_alloc = s;
	resident_changes = -1;
	set();
}

// Needs full warp
__device__ void ResidentBlock::set() {
	if (resident_changes % max_resident_changes == 0 && resident_changes != 0) {
		slab_alloc->allocateSuperBlock();
		#ifndef NDEBUG
		if(threadIdx.x % warpSize == 0)		//DEBUG
			printf(", allocateSuperBlock() called by set(), resident_changes=%d", resident_changes);
		#endif // !NDEBUG
		// resident_changes = -1;	// So it becomes 0 after a memory block is found
	}
	int global_warp_id = CEILDIV(blockDim.x, warpSize) * blockIdx.x + (threadIdx.x/warpSize);
	//unsigned memory_block_no = HashFunction::memoryblock_hash(global_warp_id, resident_changes, SuperBlock::numMemoryBlocks);
	uint32_t total_memory_blocks = slab_alloc->getNumSuperBlocks() * SuperBlock::numMemoryBlocks;
	uint32_t super_memory_block_no = HashFunction::memoryblock_hash(global_warp_id, resident_changes, total_memory_blocks);
	starting_addr = super_memory_block_no << SLAB_BITS;
	++resident_changes;
	int laneID = threadIdx.x % warpSize;
	BlockBitMap * resident_bitmap = slab_alloc->bitmaps + (starting_addr>>SLAB_BITS);
	resident_bitmap_line = resident_bitmap->bitmap[laneID];
}

__device__ Address ResidentBlock::warp_allocate() {
	//TODO remove this loop maybe
	Address allocated_address = EMPTY_ADDRESS;
	const int global_warp_id = CEILDIV(blockDim.x, warpSize) * blockIdx.x + (threadIdx.x / warpSize);
	const int max_allowed_superblock_changes = 2;
	const int max_allowed_memoryblock_changes = max_allowed_superblock_changes * max_resident_changes;
	const int max_local_rbl_changes = max_resident_changes;
	int allocator_thread_no = -1;
	int memoryblock_changes = 0, laneID = threadIdx.x % warpSize;
	for (int local_rbl_changes = 0; local_rbl_changes <= max_local_rbl_changes; ++local_rbl_changes) {		//review the loop termination condition
		int slab_no;
		while (true) {		//Review this loop
			slab_no = HashFunction::unsetbit_index(global_warp_id, local_rbl_changes, resident_bitmap_line);
			allocator_thread_no = HashFunction::unsetbit_index(global_warp_id, local_rbl_changes, ~__ballot_sync(WARP_MASK, slab_no + 1));
			if (allocator_thread_no == -1) { // All memory units are full in the memory block
				if (memoryblock_changes > max_allowed_memoryblock_changes) {
					slab_alloc->status = 1;
					__threadfence();
					int khela = 0;
					assert(khela);
					asm("trap;"); // Kills kernel with error
				}
				set();
				++memoryblock_changes;
			}
			else {
				break;
			}
		}

		if (laneID == allocator_thread_no) {
			uint32_t i = 1 << slab_no;
			auto global_memory_block_no = starting_addr >> SLAB_BITS;
			BlockBitMap* resident_bitmap = slab_alloc->bitmaps + global_memory_block_no;
			uint32_t* global_bitmap_line = resident_bitmap->bitmap + laneID;
			auto oldval = atomicOr(global_bitmap_line, i);
			resident_bitmap_line = oldval | i;
			if ((oldval & i) == 0) {
				allocated_address = starting_addr + (laneID << 5) + slab_no;
			}
		}

		__syncwarp();
		Address toreturn = __shfl_sync(WARP_MASK, allocated_address, allocator_thread_no);
		if (toreturn != EMPTY_ADDRESS) {
			return toreturn;
		}
		// TODO check for divergence on this functions return
	}
	//This means all max_local_rbl_changes attempts to allocate memory failed as the atomicCAS call kept failing
	//Terminate
	slab_alloc->status = 2;
	__threadfence();
	int mahakhela = 0;
	assert(mahakhela);
	asm("trap;");

	return EMPTY_ADDRESS; // Will never execute
}

#ifndef NDEBUG
__device__ Address ResidentBlock::warp_allocate(int * x) {		//DEBUG
	__shared__ int lrc[32][8];
	__shared__ int sn[32][8];
	__shared__ int atn[32][8];
	__shared__ uint32_t ov[32][8];
	__syncwarp();
	int warp_id_in_block = threadIdx.x / warpSize;
	for (int i = 0; i < 8; ++i)
		lrc[warp_id_in_block][i] = -1;
	//TODO remove this loop maybe
	Address allocated_address = EMPTY_ADDRESS;
	const int global_warp_id = CEILDIV(blockDim.x, warpSize) * blockIdx.x + (threadIdx.x/warpSize);
	const int max_allowed_superblock_changes = 2;
	const int max_allowed_memoryblock_changes = max_allowed_superblock_changes * max_resident_changes;
	const int max_local_rbl_changes = max_resident_changes;
	int allocator_thread_no = -1;
	int memoryblock_changes = 0, laneID = threadIdx.x % warpSize;
	for(/*int local_rbl_changes = 0*/*x = 0; /*local_rbl_changes*/*x <= max_local_rbl_changes; ++(*x) /*++local_rbl_changes*/) {		//review the loop termination condition
		int slab_no;
		auto local_rbl_changes = *x;
		while (true) {		//Review this loop
			slab_no = HashFunction::unsetbit_index(global_warp_id, local_rbl_changes, resident_bitmap_line);
			allocator_thread_no = HashFunction::unsetbit_index(global_warp_id, local_rbl_changes, ~__ballot_sync(WARP_MASK, slab_no + 1));
			if (allocator_thread_no == -1) { // All memory units are full in the memory block
				if (memoryblock_changes > max_allowed_memoryblock_changes) {
					slab_alloc->status = 1;
					__threadfence();
					int khela = 0;
					assert(khela);
					asm("trap;"); // Kills kernel with error
				}
				__syncwarp();
				if (laneID == 0)
					printf("Warp ID=%d, local_rbl_changes=%d, memoryblock_changes=%d, called set()", global_warp_id, *x, memoryblock_changes);
				set();
				if (laneID == 0)
					printf("\n");
				++memoryblock_changes;
			}
			else {
				break;
			}
		}

		if (laneID == allocator_thread_no) {
			uint32_t i = 1 << slab_no;
			auto global_memory_block_no = starting_addr >> SLAB_BITS;
			BlockBitMap* resident_bitmap = slab_alloc->bitmaps + global_memory_block_no;
			uint32_t* global_bitmap_line = resident_bitmap->bitmap + laneID;
			auto oldval = atomicOr(global_bitmap_line, i);
			resident_bitmap_line = oldval | i;
			if ((oldval & i) == 0) {
				allocated_address = starting_addr + (laneID << 5) + slab_no;
			}
			else {
				lrc[warp_id_in_block][*x] = *x;
				sn[warp_id_in_block][*x] = slab_no;
				atn[warp_id_in_block][*x] = allocator_thread_no;
				ov[warp_id_in_block][*x] = oldval;
			}
		}

		__syncwarp();
		Address toreturn = __shfl_sync(WARP_MASK, allocated_address, allocator_thread_no);
		if (toreturn != EMPTY_ADDRESS) {
			return toreturn;
		}
		// TODO check for divergence on this functions return
	}
	//This means all max_local_rbl_changes attempts to allocate memory failed as the atomicCAS call kept failing
	//Terminate
	/*slab_alloc->status = 2;
	__threadfence();
	int mahakhela = 0;
	assert(mahakhela);
	asm("trap;");*/
	__syncwarp();
	if (laneID == allocator_thread_no) {
		printf("warp_allocate() failed for Warp ID=%d. Details of each iteration:\n", global_warp_id);
		for (int i = 0; i < 8; ++i) {
			if (lrc[warp_id_in_block][i] != -1)
				printf("-> Warp ID=%d, local_rbl_changes=%d, oldval=%x, slab_no=%d, allocator_thread_no=%d\n", global_warp_id, lrc[warp_id_in_block][i], ov[warp_id_in_block][i], sn[warp_id_in_block][i], atn[warp_id_in_block][i]);
		}
		printf("-------------------------------------------------------------------------------------------------------\n");
	}
	return EMPTY_ADDRESS; // Will never execute
}
#endif // !NDEBUG
